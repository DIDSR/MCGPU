#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////////////
//
//              ****************************
//              *** MC-GPU , version 1.3 ***
//              ****************************
//                                          
//!  Definition of the CUDA GPU kernel for the simulation of x ray tracks in a voxelized geometry.
//!  This kernel has been optimized to yield a good performance in the GPU but can still be
//!  compiled in the CPU without problems. All the CUDA especific commands are enclosed in
//!  pre-processor directives that are skipped if the parameter "USING_CUDA" is not defined
//!  at compilation time.
//
//        ** DISCLAIMER **
//
// This software and documentation (the "Software") were developed at the Food and
// Drug Administration (FDA) by employees of the Federal Government in the course
// of their official duties. Pursuant to Title 17, Section 105 of the United States
// Code, this work is not subject to copyright protection and is in the public
// domain. Permission is hereby granted, free of charge, to any person obtaining a
// copy of the Software, to deal in the Software without restriction, including
// without limitation the rights to use, copy, modify, merge, publish, distribute,
// sublicense, or sell copies of the Software or derivatives, and to permit persons
// to whom the Software is furnished to do so. FDA assumes no responsibility
// whatsoever for use by other parties of the Software, its source code,
// documentation or compiled executables, and makes no guarantees, expressed or
// implied, about its quality, reliability, or any other characteristic. Further,
// use of this code in no way implies endorsement by the FDA or confers any
// advantage in regulatory decisions.  Although this software can be redistributed
// and/or modified freely, we ask that any derivative works bear some notice that
// they are derived from it, and any modified versions bear some notice that they
// have been modified.
//                                                                            
//
//!                     @file    MC-GPU_kernel_v1.3.cu
//!                     @author  Andreu Badal (Andreu.Badal-Soler@fda.hhs.gov)
//!                     @date    2012/12/12
//                       -- Original code started on:  2009/04/14
//
////////////////////////////////////////////////////////////////////////////////



////////////////////////////////////////////////////////////////////////////////
//!  Initialize the image array, ie, set all pixels to zero
//!  Essentially, this function has the same effect as the command: 
//!   "cutilSafeCall(hipMemcpy(image_device, image, image_bytes, hipMemcpyHostToDevice))";
//!  
//!  CUDA performs some initialization work the first time a GPU kernel is called.
//!  Therefore, calling a short kernel before the real particle tracking is performed
//!  may improve the accuracy of the timing measurements in the relevant kernel.
//!  
//!       @param[in,out] image   Pointer to the image array.
//!       @param[in] pixels_per_image  Number of pixels in the image (ie, elements in the array).
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__global__
void init_image_array_GPU(unsigned long long int* image, int pixels_per_image)
{
  int my_pixel = threadIdx.x + blockIdx.x*blockDim.x;
  if (my_pixel < pixels_per_image)
  {
    // -- Set the current pixel to 0 and return, avoiding overflow when more threads than pixels are used:
    image[my_pixel] = (unsigned long long int)(0);    // Initialize non-scatter image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize Compton image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize Rayleigh image
    my_pixel += pixels_per_image;                     //  (advance to next image)
    image[my_pixel] = (unsigned long long int)(0);    // Initialize multi-scatter image
  }
}

// ////////////////////////////////////////////////////////////////////////////////
// //!  Initialize the dose deposition array, ie, set all voxel doses to zero
// //!  
// //!       @param[in,out] dose   Pointer to the dose mean and sigma arrays.
// //!       @param[in] num_voxels_dose  Number of voxels in the dose ROI (ie, elements in the arrays).
// ////////////////////////////////////////////////////////////////////////////////
// __global__
// void init_dose_array_GPU(ulonglong2* voxels_Edep, int num_voxels_dose)
// {  
//   int my_voxel = threadIdx.x + blockIdx.x*blockDim.x;
//   register ulonglong2 ulonglong2_zero;
//   ulonglong2_zero.x = ulonglong2_zero.y = (unsigned long long int) 0;
//   if (my_voxel < num_voxels_dose)
//   {
//     dose[my_voxel] = ulonglong2_zero;    // Set the current voxel to (0,0) and return, avoiding overflow
//   }
// }

#endif

 
////////////////////////////////////////////////////////////////////////////////
//!  Main function to simulate x-ray tracks inside a voxelized geometry.
//!  Secondary electrons are not simulated (in photoelectric and Compton 
//!  events the energy is locally deposited).
//!
//!  The following global variables, in  the GPU __constant__ memory are used:
//!           voxel_data_CONST, 
//!           source_energy_data_CONST,
//!           detector_data_CONST, 
//!           mfp_table_data_CONST.
//!
//!       @param[in] history_batch  Particle batch number (only used in the CPU version when CUDA is disabled!, the GPU uses the built-in variable threadIdx)
//!       @param[in] num_p  Projection number in the CT simulation. This variable defines a specific angle and the corresponding source and detector will be used.
//!       @param[in] histories_per_thread   Number of histories to simulate for each call to this function (ie, for GPU thread).
//!       @param[in] seed_input   Random number generator seed (the same seed is used to initialize the two MLCGs of RANECU).
//!       @param[in] voxel_mat_dens   Pointer to the voxel densities and material vector (the voxelized geometry), stored in GPU glbal memory.
//!       @param[in] mfp_Woodcock_table    Two parameter table for the linear interpolation of the Woodcock mean free path (MFP) (stored in GPU global memory).
//!       @param[in] mfp_table_a   First element for the linear interpolation of the interaction mean free paths (stored in GPU global memory).
//!       @param[in] mfp_table_b   Second element for the linear interpolation of the interaction mean free paths (stored in GPU global memory).
//!       @param[in] rayleigh_table   Pointer to the table with the data required by the Rayleigh interaction sampling, stored in GPU global memory.
//!       @param[in] compton_table   Pointer to the table with the data required by the Compton interaction sampling, stored in GPU global memory.
//!       @param[in,out] image   Pointer to the image vector in the GPU global memory.
//!       @param[in,out] dose   Pointer to the array containing the 3D voxel dose (and its uncertainty) in the GPU global memory.
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__global__ void track_particles(int histories_per_thread,
                                int num_p,      // For a CT simulation: allocate space for up to MAX_NUM_PROJECTIONS projections.
                                int seed_input,
                                unsigned long long int* image,
                                ulonglong2* voxels_Edep,
                                float2* voxel_mat_dens,
                                float2* mfp_Woodcock_table,
                                float3* mfp_table_a,
                                float3* mfp_table_b,
                                struct rayleigh_struct* rayleigh_table,
                                struct compton_struct* compton_table,
                                struct detector_struct* detector_data_array,
                                struct source_struct* source_data_array, 
                                ulonglong2* materials_dose)
#else
           void track_particles(int history_batch,             // This variable is not required in the GPU, it uses the thread ID           
                                int histories_per_thread,
                                int num_p,
                                int seed_input,
                                unsigned long long int* image,
                                ulonglong2* voxels_Edep,
                                float2* voxel_mat_dens,
                                float2* mfp_Woodcock_table,
                                float3* mfp_table_a,
                                float3* mfp_table_b,
                                struct rayleigh_struct* rayleigh_table,
                                struct compton_struct* compton_table,
                                struct detector_struct* detector_data_array,
                                struct source_struct* source_data_array, 
                                ulonglong2* materials_dose)
#endif
{
  // -- Declare the track state variables:
  float3 position, direction;
  float energy, step, prob, randno, mfp_density, mfp_Woodcock;
  float3 mfp_table_read_a, mfp_table_read_b;
  int2 seed;
  int index;
  int material0,        // Current material, starting at 0 for 1st material
      material_old;     // Flag to mark a material or energy change
  signed char scatter_state;    // Flag for scatter images: scatter_state=0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.

  // -- Store the Compton table in shared memory from global memory:
  //    For Compton and Rayleigh the access to memory is not coherent and the caching capability do not speeds up the accesses, they actually slows down the acces to other data.
#ifdef USING_CUDA
  __shared__
#endif
  struct compton_struct cgco_SHARED;  
#ifdef USING_CUDA
  __shared__
#endif
  struct detector_struct detector_data_SHARED;
#ifdef USING_CUDA
  __shared__
#endif 
  struct source_struct source_data_SHARED;    

    
#ifdef USING_CUDA
  if (0==threadIdx.x)  // First GPU thread copies the variables to shared memory
  {
#endif

    // -Copy the current source, detector data from global to shared memory for fast access:
    source_data_SHARED    = source_data_array[num_p];      
    detector_data_SHARED  = detector_data_array[num_p];    // Copy the long array to a single instance in shared memory for the current projection
        
    // -Copy the compton data to shared memory:
    cgco_SHARED = *compton_table;
    
#ifdef USING_CUDA
  }
  __syncthreads();     // Make sure all threads will see the initialized shared variable  
#endif


  // -- Initialize the RANECU generator in a position far away from the previous history:
#ifdef USING_CUDA
  init_PRNG((threadIdx.x + blockIdx.x*blockDim.x), histories_per_thread, seed_input, &seed);   // Using a 1D block
#else
  init_PRNG(history_batch, histories_per_thread, seed_input, &seed);
#endif

  
  // -- Loop for the "histories_per_thread" particles in the current history_batch:

  for( ; histories_per_thread>0; histories_per_thread--)
  {
        //  printf("\n\n********* NEW HISTORY:  %d    [seeds: %d, %d]\n\n", histories_per_thread, seed.x, seed.y); //  fflush(stdout);  // !!Verbose!! calling printf from the GPU is possible but if multiple threads call it at the same time some output will be lost.

    int absvox = 1;
    
    // -- Call the source function to get a primary x ray:
    source(&position, &direction, &energy, &seed, &absvox, &source_data_SHARED, &detector_data_SHARED);

    scatter_state = (signed char)0;     // Reset previous scatter state: new non-scattered particle loaded

    // -- Find the current energy bin by truncation (this could be pre-calculated for a monoenergetic beam):    
    //    The initialization host code made sure that the sampled energy will always be within the tabulated energies (index never negative or too large).
#ifdef USING_CUDA
    index = __float2int_rd((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide);  // Using CUDA function to convert float to integer rounding down (towards minus infinite)
#else
    index = (int)((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide + 0.00001f);    // Adding EPSILON to truncate to INT towards minus infinite. There may be a small error for energy<=mfp_table_data_CONST.e0 but this case is irrelevant (particles will always have more energy than e0).
#endif          

  
    // -- Get the minimum mfp at the current energy using linear interpolation (Woodcock tracking):      
    {
      float2 mfp_Woodcock_read = mfp_Woodcock_table[index];   // Read the 2 parameters for the linear interpolation in a single read from global memory
      mfp_Woodcock = mfp_Woodcock_read.x + energy * mfp_Woodcock_read.y;   // Interpolated minimum MFP          
    }


    // -- Reset previous material to force a recalculation of the MFPs (negative materials are not allowed in the voxels):
    material_old  = -1;

    // *** X-ray interaction loop:
    for(;;)
    {
      
      if (absvox<0)   // !!DeBuG!!  MC-GPU_v1.3 ==> if I move this "if" above the code runs much slower!? Why???
          break;    // -- Primary particle was not pointing to the voxel region! (but may still be detected after moving in vacuum in a straight line).      


      // *** Virtual interaction loop:  // New loop structure in MC-GPU_v1.3: simulate all virtual events before sampling Compton & Rayleigh:  // !!DeBuG!!
      
      float2 matdens;
      short3 voxel_coord;    // Variable used only by DOSE TALLY

      do
      {     
        step = -(mfp_Woodcock)*logf(ranecu(&seed));   // Using the minimum MFP in the geometry for the input energy (Woodcock tracking)
          
        position.x += step*direction.x;
        position.y += step*direction.y;
        position.z += step*direction.z;

        // -- Locate the new particle in the voxel geometry:      
        absvox = locate_voxel(&position, &voxel_coord);   // Get the voxel number at the current position and the voxel coordinates (used to check if inside the dose ROI in DOSE TALLY).
        if (absvox<0)
          break;    // -- Particle escaped the voxel region! ("index" is still >0 at this moment)
          
        matdens = voxel_mat_dens[absvox];     // Get the voxel material and density in a single read from global memory
        material0 = (int)(matdens.x - 1);   // Set the current material by truncation, and set 1st material to value '0'.

        // -- Get the data for the linear interpolation of the interaction MFPs, in case the energy or material have changed:
        if (material0 != material_old)
        {
          mfp_table_read_a = mfp_table_a[index*(MAX_MATERIALS)+material0];
          mfp_table_read_b = mfp_table_b[index*(MAX_MATERIALS)+material0];
          material_old = material0;                                              // Store the new material
        }
        
        // *** Apply Woodcock tracking:
        mfp_density = mfp_Woodcock * matdens.y;
        // -- Calculate probability of delta scattering, using the total mean free path for the current material and energy (linear interpolation):
        prob = 1.0f - mfp_density * (mfp_table_read_a.x + energy * mfp_table_read_b.x);
        randno = ranecu(&seed);    // Sample uniform PRN
      }
      while (randno<prob);   // [Iterate if there is a delta scattering event]

      if (absvox<0)
        break;    // -- Particle escaped the voxel region! Break the interaction loop to call tally image.

        
      // The GPU threads will be stopped and waiting here until ALL threads have a REAL event: 

      // -- Real event takes place! Check the kind of event and sample the effects of the interaction:
      
      prob += mfp_density * (mfp_table_read_a.y + energy * mfp_table_read_b.y);    // Interpolate total Compton MFP ('y' component)
      if (randno<prob)   // [Checking Compton scattering]
      {
        // *** Compton interaction:

        //  -- Sample new direction and energy:
        double costh_Compton;
        randno = energy;     // Save temporal copy of the particle energy (variable randno not necessary until next sampling). DOSE TALLY
        
        GCOa(&energy, &costh_Compton, &material0, &seed, &cgco_SHARED);
        rotate_double(&direction, costh_Compton, /*phi=2*pi*PRN=*/ 6.28318530717958647693*ranecu_double(&seed));

        randno = energy - randno;   // Save temporal copy of the negative of the energy lost in the interaction.  DOSE TALLY

        // -- Find the new energy interval:
#ifdef USING_CUDA
        index = __float2int_rd((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide);  // Using CUDA function to convert float to integer rounding down (towards minus infinite)
#else
        index = (int)((energy-mfp_table_data_CONST.e0)*mfp_table_data_CONST.ide + 0.00001f);    // Adding EPSILON to truncate to INT
#endif          

        
        if (index>-1)  // 'index' will be negative only when the energy is below the tabulated minimum energy: particle will be then absorbed (rejected) after tallying the dose.
        {          
          // -- Get the Woodcock MFP for the new energy (energy above minimum cutoff):
          float2 mfp_Woodcock_read = mfp_Woodcock_table[index];   // Read the 2 parameters for the linear interpolation in a single read from global memory
          mfp_Woodcock = mfp_Woodcock_read.x + energy * mfp_Woodcock_read.y;   // Interpolated minimum MFP

          material_old = -2;    // Set an impossible material to force an update of the MFPs data for the nex energy interval

          // -- Update scatter state:
          if (scatter_state==(signed char)0)
            scatter_state = (signed char)1;   // Set scatter_state == 1: Compton scattered particle
          else
            scatter_state = (signed char)3;   // Set scatter_state == 3: Multi-scattered particle
        }

      }
      else
      {
        prob += mfp_density * (mfp_table_read_a.z + energy * mfp_table_read_b.z);    // Interpolate total Rayleigh MFP ('z' component)
        if (randno<prob)   // [Checking Rayleigh scattering]
        {
          // *** Rayleigh interaction:

          //  -- Sample angular deflection:
          double costh_Rayleigh;
          float pmax_current = rayleigh_table->pmax[(index+1)*MAX_MATERIALS+material0];   // Get max (ie, value for next bin?) cumul prob square form factor for Rayleigh sampling

          GRAa(&energy, &costh_Rayleigh, &material0, &pmax_current, &seed, rayleigh_table);
          rotate_double(&direction, costh_Rayleigh, /*phi=2*pi*PRN=*/ 6.28318530717958647693*ranecu_double(&seed));

          // -- Update scatter state:
          if (scatter_state==(signed char)0)
            scatter_state = (signed char)2;   // Set scatter_state == 1: Rayleigh scattered particle
          else
            scatter_state = (signed char)3;   // Set scatter_state == 3: Multi-scattered particle

        }
        else
        {
          // *** Photoelectric interaction (or pair production): mark particle for absorption after dose tally (ie, index<0)!
          randno = -energy;   // Save temporal copy of the (negative) energy deposited in the interaction (variable randno not necessary anymore).
          index = -11;       // A negative "index" marks that the particle was absorved and that it will never arrive at the detector.
        }
      }
    
      //  -- Tally the dose deposited in Compton and photoelectric interactions:
      if (randno<-0.001f)
      {
        float Edep = -1.0f*randno;   // If any energy was deposited, this variable will temporarily store the negative value of Edep.
        
        //  -- Tally the dose deposited in the current material, if enabled (ie, array allocated and not null):
        if (materials_dose!=NULL)
          tally_materials_dose(&Edep, &material0, materials_dose);    // !!tally_materials_dose!!

        //  -- Tally the energy deposited in the current voxel, if enabled (tally disabled when dose_ROI_x_max_CONST is negative). DOSE TALLY
        if (dose_ROI_x_max_CONST > -1)
          tally_voxel_energy_deposition(&Edep, &voxel_coord, voxels_Edep);

      }    

      // -- Break interaction loop for particles that have been absorved or with energy below the tabulated cutoff: particle is "absorbed" (ie, track discontinued).
      if (index<0)
        break;  
      
    }   // [Cycle the X-ray interaction loop]

    if (index>-1)
    {
      // -- Particle escaped the voxels but was not absorbed, check if it will arrive at the detector and tally its energy:
      tally_image(&energy, &position, &direction, &scatter_state, image, &source_data_SHARED, &detector_data_SHARED);
    }
  }   // [Continue with a new history]

}   // [All tracks simulated for this kernel call: return to CPU]






////////////////////////////////////////////////////////////////////////////////
//!  Tally the dose deposited in the voxels.
//!  This function is called whenever a particle suffers a Compton or photoelectric
//!  interaction. It is not necessary to call this function if the dose tally
//!  was disabled in the input file (ie, dose_ROI_x_max_CONST < 0).
//!  Electrons are not transported in MC-GPU and therefore we are approximating
//!  that the dose is equal to the KERMA (energy released by the photons alone).
//!  This approximation is acceptable when there is electronic equilibrium and when
//!  the range of the secondary electrons is shorter than the voxel size. Usually the
//!  doses will be acceptable for photon energies below 1 MeV. The dose estimates may
//!  not be accurate at the interface of low density volumes.
//!
//!  We need to use atomicAdd() in the GPU to prevent that multiple threads update the 
//!  same voxel at the same time, which would result in a lose of information.
//!  This is very improbable when using a large number of voxels but gives troubles 
//!  with a simple geometries with few voxels (in this case the atomicAdd will slow 
//!  down the code because threads will update the voxel dose secuentially).
//!
//!
//!       @param[in] Edep   Energy deposited in the interaction
//!       @param[in] voxel_coord   Voxel coordinates, needed to check if particle located inside the input region of interest (ROI)
//!       @param[out] voxels_Edep   ulonglong2 array containing the 3D voxel dose and dose^2 (ie, uncertainty) as unsigned integers scaled by SCALE_eV.
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline 
void tally_voxel_energy_deposition(float* Edep, short3* voxel_coord, ulonglong2* voxels_Edep)
{

    // !!DeBuG!! Maybe it would be faster to store a 6 element struct and save temp copy?? struct_short_int_x6_align16  dose_ROI_size = dose_ROI_size_CONST;   // Get ROI coordinates from GPU constant memory and store temporal copy
  
  if((voxel_coord->x < dose_ROI_x_min_CONST) || (voxel_coord->x > dose_ROI_x_max_CONST) ||
     (voxel_coord->y < dose_ROI_y_min_CONST) || (voxel_coord->y > dose_ROI_y_max_CONST) ||
     (voxel_coord->z < dose_ROI_z_min_CONST) || (voxel_coord->z > dose_ROI_z_max_CONST))
    {
      return;   // -- Particle outside the ROI: return without tallying anything.
    }

  // -- Particle inside the ROI: tally Edep.
  register int DX = 1 + (int)(dose_ROI_x_max_CONST - dose_ROI_x_min_CONST);
  register int num_voxel = (int)(voxel_coord->x-dose_ROI_x_min_CONST) + ((int)(voxel_coord->y-dose_ROI_y_min_CONST))*DX + ((int)(voxel_coord->z-dose_ROI_z_min_CONST))*DX*(1 + (int)(dose_ROI_y_max_CONST-dose_ROI_y_min_CONST));
  
   #ifdef USING_CUDA
     atomicAdd(&voxels_Edep[num_voxel].x, __float2ull_rn((*Edep)*SCALE_eV) );    // Energy deposited at the voxel, scaled by the factor SCALE_eV and rounded.
     atomicAdd(&voxels_Edep[num_voxel].y, __float2ull_rn((*Edep)*(*Edep)) );     // (not using SCALE_eV for std_dev to prevent overflow)           
   #else
     voxels_Edep[num_voxel].x += (unsigned long long int)((*Edep)*SCALE_eV + 0.5f);
     voxels_Edep[num_voxel].y += (unsigned long long int)((*Edep)*(*Edep) + 0.5f);
   #endif
          
  return;
}


////////////////////////////////////////////////////////////////////////////////
//!  Tally a radiographic projection image.
//!  This function is called whenever a particle escapes the voxelized volume.
//!  The code checks if the particle would arrive at the detector if it kept
//!  moving in a straight line after exiting the voxels (assuming vacuum enclosure).
//!  An ideal image formation model is implemented: each pixel counts the total energy
//!  of the x rays that enter the pixel (100% detection efficiency for any energy).
//!  The image due to primaries and different kinds of scatter is tallied separately.
//!
//!  In the GPU, and atomicAdd() function is used to make sure that multiple threads do
//!  not update the same pixel at the same time, which would result in a lose of information.
//!  Since the atomicAdd function is only available for 'unsigned long long int' data,
//!  the float pixel values are scaled by a factor "SCALE_eV" defined in the header file
//!  (eg, #define SCALE_eV 10000.0f) and stored as unsigned long long integers in main
//!  memory.
//!
//!  WARNING! If the total tallied signal (for all particles) is larger than "1.8e19/SCALE_eV",
//!    there will be a bit overflow and the value will be reset to 0 giving bogus results.
//!
//!  WARNING! The detector plane should be located outside the voxels bounding box. However, since
//!    the particles are moved outside the bbox in the last step, they could cross the detector 
//!    plane anyway. If the particles are less than 2.0 cm behind the detector, they are moved 
//!    back and detected. Therefore the detector can be a few cm inside the bbox and still work.
//!    If the Woodcock mean free path is larger than the distance from the bbox to the detector, 
//!    we may lose some particles behind the detector!
//!
//!
//!       @param[in] energy   X-ray energy
//!       @param[in] position   Particle position
//!       @param[in] direction   Particle direction (cosine vectors)
//!       @param[in] scatter_state  Flag marking primaries, single Compton, single Rayleigh or multiple scattered radiation
//!       @param[out] image   Integer array containing the image, ie, the pixel values (in tenths of meV)
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void tally_image(float* energy, float3* position, float3* direction, signed char* scatter_state, unsigned long long int* image, struct source_struct* source_data_SHARED, struct detector_struct* detector_data_SHARED)
{
  float dist_detector, rotated_position;

  if (detector_data_SHARED->rotation_flag == 1)    // -->  Initial source direction is not (0,1,0): detector has to be rotated to +Y to find the pixel number
  {
    
    // *** Skip particles not moving towards the detector. 
    //       (a) Skip particles that were deflected more than 90 deg from the original source direction (backscatter).
    //       (b) Skip particles located more than 10 cm behind the detector 
    //       (c) Skip particles for which the direction to the detector is way bigger than SDD (likely to intersect the plane outside the pixel region).
                  // !!DeBuG!! NOTE: This may give problems for big detectors very close to the source
                  
    //      !!DeBuG!! Particles located after the detector will be moved back to the surface of the detector, but 10 cm maximum!!
    //                In this way the detector can intersect the voxels bbox or be located right on the surface of the bbox: the particles will be 
    //                transported across the detector and until a little after the end of the bbox in the last step, but then moved back.
    //                This algorithm will give correct results ONLY when the detector intersects just slightly the air space around the phantom,
    //                so that the interactions after the detector are not significant (this happens sometimes using oblique beams).
    //                I could remove particles after the detector using "if (dist_detector<0.0f) return;".

    //  (a) Calculate the angle between the particle and the initial direction (dot product): reject particle if cos_angle < cos(89)==0 (angle>89deg):
    //      [Extra parenthesis are coded to suggest to the compiler the use of intrinsic multiply-add operations].

    register float cos_angle = direction->x * source_data_SHARED->direction.x +
                              (direction->y * source_data_SHARED->direction.y +
                              (direction->z * source_data_SHARED->direction.z));    
    if (cos_angle < 0.025f)
      return;  // Reject particle: Angle larger than 89 deg --> particle moving parallel to the detector or backwards towards the source!

    //   (b) Find the distance from the current particle location (likely just after the surface of the voxel bbox) to the intersection with the detector plane:
    dist_detector = ( source_data_SHARED->direction.x * (detector_data_SHARED->center.x - position->x) +
                     (source_data_SHARED->direction.y * (detector_data_SHARED->center.y - position->y) +
                     (source_data_SHARED->direction.z * (detector_data_SHARED->center.z - position->z))) ) / cos_angle;

                        
                     
// !!DeBuG!!  IF's below (used in v1.2) are not needed when checking the x ray angle:
//   if (dist_detector < -10.0f)   // !!DeBuG!! Is 10 cm enough or too much? Should I use 0? or allow any distance?
//      return;  // !!DeBuG!! Reject particles located more than 10 cm behind the detector. 10 cm was selected arbitrarily. Woodcock MFP for x-rays in bone: MFP 200 keV photons in bone ==> 4 cm.
//      
//    if (fabsf(dist_detector)>(2.1f*detector_data_CONST.sdd))          
//      return;  // Reject particle: distance to the detector plane too large, the particle is likely to travel almost parallel to the detector and will not be detected.

            
    // *** Translate the particle to the detector plane (we assume the detector is completely absorbent: 100% detection efficiency):
    position->x = position->x + dist_detector * direction->x;
    position->y = position->y + dist_detector * direction->y;
    position->z = position->z + dist_detector * direction->z;

    // *** Rotate the particle position vector to the default reference system where the detector is perpendicular to the +Y axis, then find out if the particle is located inside a pixel:
    #ifdef USING_CUDA
      rotated_position = detector_data_SHARED->rot_inv[0]*position->x + detector_data_SHARED->rot_inv[1]*position->y + detector_data_SHARED->rot_inv[2]*position->z;  // X coordinate
      int pixel_coord_x = __float2int_rd((rotated_position - detector_data_SHARED->corner_min_rotated_to_Y.x) * detector_data_SHARED->inv_pixel_size_X);    // Using CUDA intrinsic function to convert float to integer rounding down (towards minus infinite)
      if ((pixel_coord_x>-1)&&(pixel_coord_x<detector_data_SHARED->num_pixels.x))
      {
        rotated_position = detector_data_SHARED->rot_inv[6]*position->x + detector_data_SHARED->rot_inv[7]*position->y + detector_data_SHARED->rot_inv[8]*position->z;  // Z coordinate
        int pixel_coord_z = __float2int_rd((rotated_position - detector_data_SHARED->corner_min_rotated_to_Y.z) * detector_data_SHARED->inv_pixel_size_Z);
        if ((pixel_coord_z>-1)&&(pixel_coord_z<detector_data_SHARED->num_pixels.y))
        {
          // -- Particle enters the detector! Tally the particle energy in the corresponding pixel (in tenths of meV):
          //    Using a CUDA atomic function (not available for global floats yet) to read and increase the pixel value in a single instruction, blocking interferences from other threads.
          //    The offset for the primaries or scatter images are calculated considering that:
          //      scatter_state=0 for non-scattered, =1 for Compton, =2 for Rayleigh, and =3 for multiple scatter.
          atomicAdd(( image +                                                               // Pointer to beginning of image array
                    (int)(*scatter_state) * detector_data_SHARED->total_num_pixels +         // Offset to corresponding scatter image
                    (pixel_coord_x + pixel_coord_z*(detector_data_SHARED->num_pixels.x)) ),  // Offset to the corresponding pixel
                    __float2ull_rn((*energy)*SCALE_eV) );     // Energy arriving at the pixel, scaled by the factor SCALE_eV and rounded.
                                                              // The maximum unsigned long long int value is ~1.8e19:
        }
      }
    #else
      // CPU version (not using CUDA intrinsics: atomicAdd, fast type casting)
      rotated_position = detector_data_SHARED->rot_inv[0]*position->x + detector_data_SHARED->rot_inv[1]*position->y + detector_data_SHARED->rot_inv[2]*position->z;  // X coordinate
      
      float pixel_coord_x = floor((rotated_position - detector_data_SHARED->corner_min_rotated_to_Y.x)*detector_data_SHARED->inv_pixel_size_X);   // Using float+floor instead of INT to avoid truncation errors for positive and negative values
      if ( (pixel_coord_x>-0.1f) && (pixel_coord_x<(detector_data_SHARED->num_pixels.x-0.1f)) )    // Rejecting values negative or bigger than the image size
      {
        rotated_position = detector_data_SHARED->rot_inv[6]*position->x + detector_data_SHARED->rot_inv[7]*position->y + detector_data_SHARED->rot_inv[8]*position->z;  // Z coordinate
        float pixel_coord_z = floor((rotated_position - detector_data_SHARED->corner_min_rotated_to_Y.z)*detector_data_SHARED->inv_pixel_size_Z);
        if ( (pixel_coord_z>-0.1f) && (pixel_coord_z<(detector_data_SHARED->num_pixels.y-0.1f)) )
          image[(int)(((float)*scatter_state)*detector_data_SHARED->total_num_pixels + pixel_coord_x + pixel_coord_z*detector_data_SHARED->num_pixels.x  +  0.0001f)]
             += (unsigned long long int)((*energy)*SCALE_eV + 0.5f);   // Tally the particle energy in the pixel. This instruction is not thread-safe, but it is ok in sequential CPU code.          
      }
    #endif
  }
  else  // (detector_data_SHARED->rotation_flag != 1) -->  Initial source direction is (0,1,0): pixel number and distance can be found easily
  {  
    if (direction->y < 0.0001f)
      return;  // *** Reject particles not moving towards the detector plane at +Y.

    dist_detector = (detector_data_SHARED->center.y - position->y)/(direction->y);  // Distance to the intersection with the detector at +Y.
  
      // !!DeBuG!! IF below (v1.2) not needed when checking the angle
      //     if (dist_detector>(2.1f*detector_data_SHARED->sdd)) return;  
     
    
    #ifdef USING_CUDA
    int pixel_coord_x = __float2int_rd((position->x + dist_detector*direction->x - detector_data_SHARED->corner_min_rotated_to_Y.x)*detector_data_SHARED->inv_pixel_size_X);
    if ((pixel_coord_x>-1)&&(pixel_coord_x<detector_data_SHARED->num_pixels.x))
    {
      int pixel_coord_z = __float2int_rd((position->z + dist_detector*direction->z - detector_data_SHARED->corner_min_rotated_to_Y.z)*detector_data_SHARED->inv_pixel_size_Z);
      if ((pixel_coord_z>-1)&&(pixel_coord_z<detector_data_SHARED->num_pixels.y))
        atomicAdd( ( image +                                                                // Pointer to beginning of image array
                     (int)(*scatter_state) * detector_data_SHARED->total_num_pixels +         // Offset to corresponding scatter image
                     (pixel_coord_x + pixel_coord_z*(detector_data_SHARED->num_pixels.x)) ),  // Offset to the corresponding pixel
                   __float2ull_rn((*energy)*SCALE_eV) );    // Energy arriving at the pixel, scaled by the factor SCALE_eV and rounded.
    }
    #else

    // --Calculate the pixel the xray enters, truncating towards minus infinite and making sure the conversion to int is safe:
    float pixel_coord_x = floor((position->x + dist_detector*direction->x - detector_data_SHARED->corner_min_rotated_to_Y.x)*detector_data_SHARED->inv_pixel_size_X);

    if ( (pixel_coord_x>-0.1f) && (pixel_coord_x<(detector_data_SHARED->num_pixels.x-0.1f)) )
    {
      float pixel_coord_z = floor((position->z + dist_detector*direction->z - detector_data_SHARED->corner_min_rotated_to_Y.z)*detector_data_SHARED->inv_pixel_size_Z);
      if ( (pixel_coord_z>-0.1f) && (pixel_coord_z<(detector_data_SHARED->num_pixels.y-0.1f)) )
        image[(int)(((float)*scatter_state)*detector_data_SHARED->total_num_pixels + pixel_coord_x + pixel_coord_z*detector_data_SHARED->num_pixels.x  +  0.0001f)]
           += (unsigned long long int)((*energy)*SCALE_eV + 0.5f);    // Truncate the pixel number to INT and round the energy value
    }
    #endif
  }

}



////////////////////////////////////////////////////////////////////////////////
//!  Source that creates primary x rays, according to the defined source model.
//!  The particles are automatically moved to the surface of the voxel bounding box,
//!  to start the tracking inside a real material. If the sampled particle do not
//!  enter the voxels, it is init in the focal spot and the main program will check
//!  if it arrives at the detector or not.
//!
//!       @param[in] source_data   Structure describing the source.
//!       @param[in] source_energy_data_CONST   Global variable in constant memory space describing the source energy spectrum.
//!       @param[out] position   Initial particle position (particle transported inside the voxel bbox).
//!       @param[out] direction   Sampled particle direction (cosine vectors).
//!       @param[out] energy   Sampled energy of the new x ray.
//!       @param[in] seed   Current seed of the random number generator, requiered to sample the movement direction.
//!       @param[out] absvox   Set to <0 if primary particle will not cross the voxels, not changed otherwise (>0).
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void source(float3* position, float3* direction, float* energy, int2* seed, int* absvox, struct source_struct* source_data_SHARED, struct detector_struct* detector_data_SHARED)
{
  // *** Sample the initial x-ray energy following the input energy spectrum using the Walker aliasing algorithm from PENELOPE:
      // The following code is equivalent to calling the function "seeki_walker": int sampled_bin = seeki_walker(source_data_CONST.espc_cutoff, source_data_CONST.espc_alias, ranecu(seed), source_data_CONST.num_bins_espc);      
  int sampled_bin;
  float RN = ranecu(seed) * source_energy_data_CONST.num_bins_espc;    // Find initial interval (array starting at 0):   
  #ifdef USING_CUDA
    int int_part = __float2int_rd(RN);                          //   -- Integer part (round down)
  #else
    int int_part = (int)(RN);
  #endif
  float fraction_part = RN - ((float)int_part);                 //   -- Fractional part
  if (fraction_part < source_energy_data_CONST.espc_cutoff[int_part])  // Check if we are in the aliased part
    sampled_bin = int_part;                                     // Below the cutoff: return current value
  else
    sampled_bin = (int)source_energy_data_CONST.espc_alias[int_part];  // Above the cutoff: return alias
  
  // Linear interpolation of the final energy within the sampled energy bin:
  *energy = source_energy_data_CONST.espc[sampled_bin] + ranecu(seed) * (source_energy_data_CONST.espc[sampled_bin+1] - source_energy_data_CONST.espc[sampled_bin]);   
      
 
   // *** Sample the initial direction:
   
  do   //  Iterate sampling if the sampled direction is not acceptable to get a square field at the given phi (rejection sampling): force square field for any phi!!
  {
    //     Using the algorithm used in PENMAIN.f, from penelope 2008 (by F. Salvat).
    direction->z = source_data_SHARED->cos_theta_low + ranecu(seed)*source_data_SHARED->D_cos_theta;     // direction->z = w = cos(theta_sampled)
    register float phi_sampled = source_data_SHARED->phi_low + ranecu(seed)*source_data_SHARED->D_phi;
    register float sin_theta_sampled = sqrtf(1.0f - direction->z*direction->z);
    float sinphi_sampled, cosphi_sampled;
    
    #ifdef USING_CUDA
      sincos(phi_sampled, &sinphi_sampled,&cosphi_sampled);    // Calculate the SIN and COS at the same time.
    #else
      sinphi_sampled = sin(phi_sampled);   // Some CPU compilers will be able to use "sincos", but let's be safe.
      cosphi_sampled = cos(phi_sampled);
    #endif       
    
    direction->y = sin_theta_sampled * sinphi_sampled;
    direction->x = sin_theta_sampled * cosphi_sampled;
  }
  while( fabsf(direction->z/(direction->y+1.0e-7f)) > source_data_SHARED->max_height_at_y1cm );  // !!DeBuG!! Force square field for any phi by rejection sampling!! Is it necessary to use the "+1.0e-7f" to prevent possible division by zero???
    

  if (detector_data_SHARED->rotation_flag == 1)
  {
    // --Initial beam not pointing to (0,1,0), apply rotation:
    register float direction_x_tmp = direction->x;
    register float direction_y_tmp = direction->y;
    direction->x = source_data_SHARED->rot_fan[0]*direction_x_tmp + source_data_SHARED->rot_fan[1]*direction_y_tmp + source_data_SHARED->rot_fan[2]*direction->z;
    direction->y = source_data_SHARED->rot_fan[3]*direction_x_tmp + source_data_SHARED->rot_fan[4]*direction_y_tmp + source_data_SHARED->rot_fan[5]*direction->z;
    direction->z = source_data_SHARED->rot_fan[6]*direction_x_tmp + source_data_SHARED->rot_fan[7]*direction_y_tmp + source_data_SHARED->rot_fan[8]*direction->z;
  }

  // Initialize x ray position at focal spot before translation into bbox. Particle stays in focal spot if no interaction found:
  position->x = source_data_SHARED->position.x;
  position->y = source_data_SHARED->position.y;
  position->z = source_data_SHARED->position.z;
      
  move_to_bbox(position, direction, voxel_data_CONST.size_bbox, absvox);  // Move the particle inside the voxel bounding box.
}



////////////////////////////////////////////////////////////////////////////////
//!  Functions that moves a particle inside the voxelized geometry bounding box.
//!  An EPSILON distance is added to make sure the particles will be clearly inside the bbox, 
//!  not exactly on the surface. 
//!
//!  This algorithm makes the following assumtions:
//!     - The back lower vertex of the voxel bounding box is always located at the origin: (x0,y0,z0)=(0,0,0).
//!     - The initial value of "position" corresponds to the focal spot location.
//!     - When a ray is not pointing towards the bbox plane that it should cross according to the sign of the direction,
//!       I assign a distance to the intersection =0 instead of the real negative distance. The wall that will be 
//!       crossed to enter the bbox is always the furthest and therefore a 0 distance will never be used except
//!       in the case of a ray starting inside the bbox or outside the bbox and not pointing to any of the 3 planes. 
//!       In this situation the ray will be transported a 0 distance, meaning that it will stay at the focal spot.
//!
//!  (Interesting information on ray-box intersection: http://tog.acm.org/resources/GraphicsGems/gems/RayBox.c)
//!
//!       @param[in,out] position Particle position: initially set to the focal spot, returned transported inside the voxel bbox.
//!       @param[out] direction   Sampled particle direction (cosine vectors).
//!       @param[out] intersection_flag   Set to <0 if particle outside bbox and will not cross the voxels, not changed otherwise.
//!       @param[out] size_bbox   Size of the bounding box.
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void move_to_bbox(float3* position, float3* direction, float3 size_bbox, int* intersection_flag)
{
  float dist_y, dist_x, dist_z;

  // -Distance to the nearest Y plane:
  if ((direction->y) > EPS_SOURCE)   // Moving to +Y: check distance to y=0 plane
  {
    // Check Y=0 (bbox wall):
    if (position->y > 0.0f)  // The input position must correspond to the focal spot => position->y == source_data_CONST.position[*num_p].y
      dist_y = 0.0f;  // No intersection with this plane: particle inside or past the box  
          // The actual distance would be negative but we set it to 0 bc we will not move the particle if no intersection exist.
    else
      dist_y = EPS_SOURCE + (-position->y)/(direction->y);    // dist_y > 0 for sure in this case
  }
  else if ((direction->y) < NEG_EPS_SOURCE)
  {
    // Check Y=voxel_data_CONST.size_bbox.y:
    if (position->y < size_bbox.y)
      dist_y = 0.0f;  // No intersection with this plane
    else
      dist_y = EPS_SOURCE + (size_bbox.y - position->y)/(direction->y);    // dist_y > 0 for sure in this case
  }
  else   // (direction->y)~0
    dist_y = NEG_INF;   // Particle moving parallel to the plane: no interaction possible (set impossible negative dist = -INFINITE)

  // -Distance to the nearest X plane:
  if ((direction->x) > EPS_SOURCE)
  {
    // Check X=0:
    if (position->x > 0.0f)
      dist_x = 0.0f;
    else  
      dist_x = EPS_SOURCE + (-position->x)/(direction->x);    // dist_x > 0 for sure in this case
  }
  else if ((direction->x) < NEG_EPS_SOURCE)
  {
    // Check X=voxel_data_CONST.size_bbox.x:
    if (position->x < size_bbox.x)
      dist_x = 0.0f;
    else  
      dist_x = EPS_SOURCE + (size_bbox.x - position->x)/(direction->x);    // dist_x > 0 for sure in this case
  }
  else
    dist_x = NEG_INF;

  // -Distance to the nearest Z plane:
  if ((direction->z) > EPS_SOURCE)
  {
    // Check Z=0:
    if (position->z > 0.0f)
      dist_z = 0.0f;
    else
      dist_z = EPS_SOURCE + (-position->z)/(direction->z);    // dist_z > 0 for sure in this case
  }
  else if ((direction->z) < NEG_EPS_SOURCE)
  {
    // Check Z=voxel_data_CONST.size_bbox.z:
    if (position->z < size_bbox.z)
      dist_z = 0.0f;
    else
      dist_z = EPS_SOURCE + (size_bbox.z - position->z)/(direction->z);    // dist_z > 0 for sure in this case
  }
  else
    dist_z = NEG_INF;

  
  // -- Find the longest distance plane, which is the one that has to be crossed to enter the bbox.
  //    Storing the maximum distance in variable "dist_z". Distance will be =0 if no intersection exists or 
  //    if the x ray is already inside the bbox.
  if ( (dist_y>dist_x) && (dist_y>dist_z) )
    dist_z = dist_y;      // dist_z == dist_max 
  else if (dist_x>dist_z)
    dist_z = dist_x;
// else
//   dist_max = dist_z;
    
  // -- Move particle from the focal spot (current location) to the bbox wall surface (slightly inside):
  position->x += dist_z * direction->x;
  position->y += dist_z * direction->y;
  position->z += dist_z * direction->z;      
  
  // Check if the new position is outside the bbox. If true, the particle must be moved back to the focal spot location:
  if ( (position->x < 0.0f) || (position->x > size_bbox.x) || 
       (position->y < 0.0f) || (position->y > size_bbox.y) || 
       (position->z < 0.0f) || (position->z > size_bbox.z) )
  {
    position->x -= dist_z * direction->x;  // Reject new position undoing the previous translation
    position->y -= dist_z * direction->y;
    position->z -= dist_z * direction->z;
    (*intersection_flag) = -111;  // Particle outside the bbox AND not pointing to the bbox: set absvox<0 to skip interaction sampling.
  }
}


////////////////////////////////////////////////////////////////////////////////


//!  Upper limit of the number of random values sampled in a single track.
#define  LEAP_DISTANCE     256
//!  Multipliers and moduli for the two MLCG in RANECU.
#define  a1_RANECU       40014
#define  m1_RANECU  2147483563
#define  a2_RANECU       40692
#define  m2_RANECU  2147483399
////////////////////////////////////////////////////////////////////////////////
//! Initialize the pseudo-random number generator (PRNG) RANECU to a position
//! far away from the previous history (leap frog technique).
//!
//! Each calculated seed initiates a consecutive and disjoint sequence of
//! pseudo-random numbers with length LEAP_DISTANCE, that can be used to
//! in a parallel simulation (Sequence Splitting parallelization method).
//! The basic equation behind the algorithm is:
//!    S(i+j) = (a**j * S(i)) MOD m = [(a**j MOD m)*S(i)] MOD m  ,
//! which is described in:
//!   P L'Ecuyer, Commun. ACM 31 (1988) p.742
//!
//! This function has been adapted from "seedsMLCG.f", see:
//!   A Badal and J Sempau, Computer Physics Communications 175 (2006) p. 440-450
//!
//!       @param[in] history   Particle bach number.
//!       @param[in] seed_input   Initial PRNG seed input (used to initiate both MLCGs in RANECU).
//!       @param[out] seed   Initial PRNG seeds for the present history.
//!
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int2* seed)
{
  // -- Move the RANECU generator to a unique position for the current batch of histories:
  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
  // **** 1st MLCG:
  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
  int y = 1;
  int z = a1_RANECU;
  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
  for(;;)
  {
    // (A2) Halve n, and store the integer part and the residue
    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
    {
      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);  
      y = abMODm(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
      if (0==leap) break;         // (A4) leap==0? ==> finish
    }
    else           // (leap is even)
    {
      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
    }
    z = abMODm(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
  }
  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
  //         S_i = abMODm(m,S_i,AjMODm)
  seed->x = abMODm(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
  y = 1;
  z = a2_RANECU;
  for(;;)
  {
    // (A2) Halve n, and store the integer part and the residue
    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
    {
      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);
      y = abMODm(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
      if (0==leap) break;         // (A4) leap==0? ==> finish
    }
    else           // (leap is even)
    {
      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
    }
    z = abMODm(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
  }
  // AjMODm2 = y;
  seed->y = abMODm(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed
}



/////////////////////////////////////////////////////////////////////
//!  Calculate "(a1*a2) MOD m" with 32-bit integers and avoiding
//!  the possible overflow, using the Russian Peasant approach
//!  modulo m and the approximate factoring method, as described
//!  in:  L'Ecuyer and Cote, ACM Trans. Math. Soft. 17 (1991).
//!
//!  This function has been adapted from "seedsMLCG.f", see: 
//!  Badal and Sempau, Computer Physics Communications 175 (2006)
//!
//!       @param[in] m,a,s  MLCG parameters
//!       @return   (a1*a2) MOD m   
//
//    Input:          0 < a1 < m                                  
//                    0 < a2 < m                                  
//
//    Return value:  (a1*a2) MOD m                                
//
/////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__ __host__    // Function will be callable from host and also from device
#endif
inline int abMODm(int m, int a, int s)
{
  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)
  int q, k;
  int p = -m;            // p is always negative to avoid overflow when adding

  // ** Apply the Russian peasant method until "a =< 32768":
  while (a>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
  {
    if (0!=(a&1))        // Store 's' when 'a' is odd     Equivalent code:   if (1==(a%2))
    {
      p += s;
      if (p>0) p -= m;
    }
    a >>= 1;             // Half a (move bits 1 position right)   Equivalent code: a = a/2;
    s = (s-m) + s;       // Double s (MOD m)
    if (s<0) s += m;     // (s is always positive)
  }

  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
  q = (int) m / a;
  k = (int) s / q;
  s = a*(s-k*q)-k*(m-q*a);
  while (s<0)
    s += m;

  // ** Compute the final result:
  p += s;
  if (p<0) p += m;

  return p;
}



////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a float value
//! (single precision version).
//!
//!       @param[in,out] seed   PRNG seed (seed kept in the calling function and updated here).
//!       @return   PRN double value in the open interval (0,1)
//!
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__ 
#endif
inline float ranecu(int2* seed)
{
  int i1 = (int)(seed->x/53668);
  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

  int i2 = (int)(seed->y/52774);
  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

  if (seed->x < 0) seed->x += 2147483563;
  if (seed->y < 0) seed->y += 2147483399;

  i2 = seed->x-seed->y;
  if (i2 < 1) i2 += 2147483562;


#ifdef USING_CUDA
  return (__int2float_rn(i2)*4.65661305739e-10f);        // 4.65661305739e-10 == 1/2147483563
#else
  return ((float)(i2)*4.65661305739e-10f);          
#endif

}


////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a double value.
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__ 
#endif
inline double ranecu_double(int2* seed)
{
  int i1 = (int)(seed->x/53668);
  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

  int i2 = (int)(seed->y/52774);
  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

  if (seed->x < 0) seed->x += 2147483563;
  if (seed->y < 0) seed->y += 2147483399;

  i2 = seed->x-seed->y;
  if (i2 < 1) i2 += 2147483562;

#ifdef USING_CUDA
  return (__int2double_rn(i2)*4.6566130573917692e-10);
#else
  return ((double)(i2)*4.6566130573917692e-10);
#endif

}



////////////////////////////////////////////////////////////////////////////////
//! Find the voxel that contains the current position.
//! Report the voxel absolute index and the x,y,z indices.
//! The structure containing the voxel number and size is read from CONSTANT memory.
//!
//!       @param[in] position   Particle position
//!       @param[out] voxel_coord   Pointer to three integer values (short3*) that will store the x,y and z voxel indices.
//!       @return   Returns "absvox", the voxel number where the particle is
//!                 located (negative if position outside the voxel bbox).
//!
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline int locate_voxel(float3* position, short3* voxel_coord)
{

  if ( (position->y < EPS_SOURCE) || (position->y > (voxel_data_CONST.size_bbox.y - EPS_SOURCE)) ||
       (position->x < EPS_SOURCE) || (position->x > (voxel_data_CONST.size_bbox.x - EPS_SOURCE)) ||
       (position->z < EPS_SOURCE) || (position->z > (voxel_data_CONST.size_bbox.z - EPS_SOURCE)) )
  {
    // -- Particle escaped the voxelized geometry (using EPS_SOURCE to avoid numerical precision errors):      
     return -1;
  }
 
  // -- Particle inside the voxelized geometry, find current voxel:
  //    The truncation from float to integer could give troubles for negative coordinates but this will never happen thanks to the IF at the begining of this function.
  //    (no need to use the CUDA function to convert float to integer rounding down (towards minus infinite): __float2int_rd)
  
  register int voxel_coord_x, voxel_coord_y, voxel_coord_z;
#ifdef USING_CUDA
  voxel_coord_x = __float2int_rd(position->x * voxel_data_CONST.inv_voxel_size.x);  
  voxel_coord_y = __float2int_rd(position->y * voxel_data_CONST.inv_voxel_size.y);
  voxel_coord_z = __float2int_rd(position->z * voxel_data_CONST.inv_voxel_size.z);
#else
  voxel_coord_x = (int)(position->x * voxel_data_CONST.inv_voxel_size.x);     
  voxel_coord_y = (int)(position->y * voxel_data_CONST.inv_voxel_size.y);
  voxel_coord_z = (int)(position->z * voxel_data_CONST.inv_voxel_size.z);
#endif

  // Output the voxel coordinates as short int (2 bytes) instead of int (4 bytes) to save registers; avoid type castings in the calculation of the return value.
  voxel_coord->x = (short int) voxel_coord_x;
  voxel_coord->y = (short int) voxel_coord_y;
  voxel_coord->z = (short int) voxel_coord_z;
  
  return (voxel_coord_x + voxel_coord_y*(voxel_data_CONST.num_voxels.x) + voxel_coord_z*(voxel_data_CONST.num_voxels.x)*(voxel_data_CONST.num_voxels.y));  
}



//////////////////////////////////////////////////////////////////////
//!   Rotates a vector; the rotation is specified by giving
//!   the polar and azimuthal angles in the "self-frame", as
//!   determined by the vector to be rotated.
//!   This function is a literal translation from Fortran to C of
//!   PENELOPE (v. 2006) subroutine "DIRECT".
//!
//!    @param[in,out]  (u,v,w)  input vector (=d) in the lab. frame; returns the rotated vector components in the lab. frame
//!    @param[in]  costh  cos(theta), angle between d before and after turn
//!    @param[in]  phi  azimuthal angle (rad) turned by d in its self-frame
//
//    Output:
//      (u,v,w) -> rotated vector components in the lab. frame
//
//    Comments:
//      -> (u,v,w) should have norm=1 on input; if not, it is
//         renormalized on output, provided norm>0.
//      -> The algorithm is based on considering the turned vector
//         d' expressed in the self-frame S',
//           d' = (sin(th)cos(ph), sin(th)sin(ph), cos(th))
//         and then apply a change of frame from S' to the lab
//         frame. S' is defined as having its z' axis coincident
//         with d, its y' axis perpendicular to z and z' and its
//         x' axis equal to y'*z'. The matrix of the change is then
//                   / uv/rho    -v/rho    u \
//          S ->lab: | vw/rho     u/rho    v |  , rho=(u^2+v^2)^0.5
//                   \ -rho       0        w /
//      -> When rho=0 (w=1 or -1) z and z' are parallel and the y'
//         axis cannot be defined in this way. Instead y' is set to
//         y and therefore either x'=x (if w=1) or x'=-x (w=-1)
//////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void rotate_double(float3* direction, double costh, double phi)   // !!DeBuG!! The direction vector is single precision but the rotation is performed in doule precision for increased accuracy.
{
  double DXY, NORM, cosphi, sinphi, SDT;
  DXY = direction->x*direction->x + direction->y*direction->y;
  
#ifdef USING_CUDA
  sincos(phi, &sinphi,&cosphi);   // Calculate the SIN and COS at the same time.
#else
  sinphi = sin(phi);   // Some CPU compilers will be able to use "sincos", but let's be safe.
  cosphi = cos(phi);
#endif   

  // ****  Ensure normalisation
  NORM = DXY + direction->z*direction->z;     // !!DeBuG!! Check if it is really necessary to renormalize in a real simulation!!
  if (fabs(NORM-1.0)>1.0e-14)
  {
    NORM = 1.0/sqrt(NORM);
    direction->x = NORM*direction->x;
    direction->y = NORM*direction->y;
    direction->z = NORM*direction->z;
    DXY = direction->x*direction->x + direction->y*direction->y;
  }
  if (DXY>1.0e-28)
  {
    SDT = sqrt((1.0-costh*costh)/DXY);
    float direction_x_in = direction->x;
    direction->x = direction->x*costh + SDT*(direction_x_in*direction->z*cosphi-direction->y*sinphi);
    direction->y = direction->y*costh+SDT*(direction->y*direction->z*cosphi+direction_x_in*sinphi);
    direction->z = direction->z*costh-DXY*SDT*cosphi;
  }
  else
  {
    SDT = sqrt(1.0-costh*costh);
    direction->y = SDT*sinphi;
    if (direction->z>0.0)
    {
      direction->x = SDT*cosphi;
      direction->z = costh;
    }
    else
    {
      direction->x =-SDT*cosphi;
      direction->z =-costh;
    }
  }
}


//////////////////////////////////////////////////////////////////////


//  ***********************************************************************
//  *   Translation of PENELOPE's "SUBROUTINE GRAa" from FORTRAN77 to C   *
//  ***********************************************************************
//!  Sample a Rayleigh interaction using the sampling algorithm
//!  used in PENELOPE 2006.
//!
//!       @param[in] energy   Particle energy (not modified with Rayleigh)
//!       @param[out] costh_Rayleigh   Cosine of the angular deflection
//!       @param[in] material  Current voxel material
//
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//  C  PENELOPE/PENGEOM (version 2006)                                     C
//  C    Copyright (c) 2001-2006                                           C
//  C    Universitat de Barcelona                                          C
//  C  Permission to use, copy, modify, distribute and sell this software  C
//  C  and its documentation for any purpose is hereby granted without     C
//  C  fee, provided that the above copyright notice appears in all        C
//  C  copies and that both that copyright notice and this permission      C
//  C  notice appear in all supporting documentation. The Universitat de   C
//  C  Barcelona makes no representations about the suitability of this    C
//  C  software for any purpose. It is provided "as is" without express    C
//  C  or implied warranty.                                                C
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline void GRAa(float *energy, double *costh_Rayleigh, int *mat, float *pmax_current, int2 *seed, struct rayleigh_struct* cgra)
{
/*  ****  Energy grid and interpolation constants for the current energy. */
    double  xmax = ((double)*energy) * 8.065535669099010e-5;       // 8.065535669099010e-5 == 2.0*20.6074/510998.918
    double x2max = min_value( (xmax*xmax) , ((double)cgra->xco[(*mat+1)*NP_RAYLEIGH - 1]) );   // Get the last tabulated value of xco for this mat
    
    if (xmax < 0.01)
    {
       do
       {
          *costh_Rayleigh = 1.0 - ranecu_double(seed) * 2.0;
       }
       while ( ranecu_double(seed) > (((*costh_Rayleigh)*(*costh_Rayleigh)+1.0)*0.5) );
       return;
    }

    for(;;)    // (Loop will iterate everytime the sampled value is rejected or above maximum)
    {
      double ru = ranecu_double(seed) * (double)(*pmax_current);    // Pmax for the current energy is entered as a parameter
 
/*  ****  Selection of the interval  (binary search within pre-calculated limits). */
      int itn = (int)(ru * (NP_RAYLEIGH-1));     // 'itn' will never reach the last interval 'NP_RAYLEIGH-1', but this is how RITA is implemented in PENELOPE
      int i__ = (int)cgra->itlco[itn + (*mat)*NP_RAYLEIGH];
      int j   = (int)cgra->ituco[itn + (*mat)*NP_RAYLEIGH];
      
      if ((j - i__) > 1)
      {
        do
        {
          register int k = (i__ + j)>>1;     // >>1 == /2 
          if (ru > cgra->pco[k -1 + (*mat)*NP_RAYLEIGH])
            i__ = k;
          else
            j = k;
        }
        while ((j - i__) > 1);
      }
       
/*  ****  Sampling from the rational inverse cumulative distribution. */
      int index = i__ - 1 + (*mat)*NP_RAYLEIGH;

      double rr = ru - cgra->pco[index];
      double xx;
      if (rr > 1e-16)
      {      
        double d__ = (double)(cgra->pco[index+1] - cgra->pco[index]);
        float aco_index = cgra->aco[index], bco_index = cgra->bco[index], xco_index = cgra->xco[index];   // Avoid multiple accesses to the same global variable

        xx = (double)xco_index + (double)(aco_index + 1.0f + bco_index)* d__* rr / (d__*d__ + (aco_index*d__ + bco_index*rr) * rr) * (double)(cgra->xco[index+1] - xco_index);
        
      }
      else
      {
        xx = cgra->xco[index];
      }
      
      if (xx < x2max)
      {
        // Sampled value below maximum possible value:
        *costh_Rayleigh = 1.0 - 2.0 * xx / x2max;   // !!DeBuG!! costh_Rayleigh in double precision, but not all intermediate steps are!?
        /*  ****  Rejection: */    
        if (ranecu_double(seed) < (((*costh_Rayleigh)*(*costh_Rayleigh) + 1.0)*0.5))
          break;   // Sample value not rejected! break loop and return.
      }
    }
} /* graa */



//////////////////////////////////////////////////////////////////////////


//  ***********************************************************************
//  *   Translation of PENELOPE's "SUBROUTINE GCOa"  from FORTRAN77 to C  *
//  ********************************************************************* *
//!  Random sampling of incoherent (Compton) scattering of photons, using 
//!  the sampling algorithm from PENELOPE 2006:
//!    Relativistic impulse approximation with analytical one-electron Compton profiles

// !!DeBuG!!  In penelope, Doppler broadening is not used for E greater than 5 MeV.
//            We don't use it in GPU to reduce the lines of code and prevent using COMMON/compos/ZT(M)

//!       @param[in,out] energy   incident and final photon energy (eV)
//!       @param[out] costh_Compton   cosine of the polar scattering angle
//!       @param[in] material   Current voxel material
//!       @param[in] seed   RANECU PRNG seed
//
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//  C  PENELOPE/PENGEOM (version 2006)                                     C
//  C    Copyright (c) 2001-2006                                           C
//  C    Universitat de Barcelona                                          C
//  C  Permission to use, copy, modify, distribute and sell this software  C
//  C  and its documentation for any purpose is hereby granted without     C
//  C  fee, provided that the above copyright notice appears in all        C
//  C  copies and that both that copyright notice and this permission      C
//  C  notice appear in all supporting documentation. The Universitat de   C
//  C  Barcelona makes no representations about the suitability of this    C
//  C  software for any purpose. It is provided "as is" without express    C
//  C  or implied warranty.                                                C
//  CCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCCC
//
//  ************************************************************************

#ifdef USING_CUDA
__device__
#endif
inline void GCOa(float *energy, double *costh_Compton, int *mat, int2 *seed, struct compton_struct* cgco_SHARED)
{
    float s, a1, s0, af, ek, ek2, ek3, tau, pzomc, taumin;
    float rn[MAX_SHELLS];
    double cdt1;

     // Some variables used in PENELOPE have been eliminated to save register: float aux, taum2, fpzmax, a, a2, ek1 ,rni, xqc, fpz, pac[MAX_SHELLS];

    int i__;
    int my_noscco = cgco_SHARED->noscco[*mat];    // Store the number of oscillators for the input material in a local variable
    
#ifndef USING_CUDA
    static int warning_flag_1 = -1, warning_flag_2 = -1, warning_flag_3 = -1;    // Write warnings for the CPU code, but only once.  !!DeBuG!!
#endif

    ek = *energy * 1.956951306108245e-6f;    // (1.956951306108245e-6 == 1.0/510998.918)
    ek2 = ek * 2.f + 1.f;
    ek3 = ek * ek;
    // ek1 = ek3 - ek2 - 1.;
    taumin = 1.f / ek2;
    // taum2 = taumin * taumin;
    a1 = logf(ek2);
    // a2 = a1 + ek * 2. * (ek + 1.) * taum2;    // a2 was used only once, code moved below


/*  ****  Incoherent scattering function for theta=PI. */

    s0 = 0.0f;
    for (i__ = 0; i__ < my_noscco; i__++)
    {
       register float temp = cgco_SHARED->uico[*mat + i__*MAX_MATERIALS];
       if (temp < *energy)
       {
         register float aux = *energy * (*energy - temp) * 2.f;
         #ifdef USING_CUDA
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) * rsqrtf(aux + aux + temp * temp) * 1.956951306108245e-6f;
             // 1.956951306108245e-6 = 1.0/510998.918f   // Version using the reciprocal of sqrt in CUDA: faster and more accurate!!
         #else
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) / (sqrtf(aux + aux + temp * temp) * 510998.918f);
         #endif
         if (pzomc > 0.0f)
           temp = (0.707106781186545f+pzomc*1.4142135623731f) * (0.707106781186545f+pzomc*1.4142135623731f);
         else
           temp = (0.707106781186545f-pzomc*1.4142135623731f) * (0.707106781186545f-pzomc*1.4142135623731f);

         temp = 0.5f * expf(0.5f - temp);    // Calculate EXP outside the IF to avoid branching

         if (pzomc > 0.0f)
            temp = 1.0f - temp;
                                
         s0 += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * temp;
       }
    }
            
/*  ****  Sampling tau. */
    do
    {
      if (ranecu(seed)*/*a2=*/(a1+2.*ek*(ek+1.f)*taumin*taumin) < a1)
      { 
        tau = powf(taumin, ranecu(seed));    // !!DeBuG!!  "powf()" has a big error (7 ULP), the double version has only 2!! 
      }
      else
      {
        tau = sqrtf(1.f + ranecu(seed) * (taumin * taumin - 1.f));
      }

      cdt1 = (double)(1.f-tau) / (((double)tau)*((double)*energy)*1.956951306108245e-6);    // !!DeBuG!! The sampled COS will be double precision, but TAU is not!!!

      if (cdt1 > 2.0) cdt1 = 1.99999999;   // !!DeBuG!! Make sure that precision error in POW, SQRT never gives cdt1>2 ==> costh_Compton<-1
      
  /*  ****  Incoherent scattering function. */
      s = 0.0f;
      for (i__ = 0; i__ < my_noscco; i__++)
      {
        register float temp = cgco_SHARED->uico[*mat + i__*MAX_MATERIALS];
        if (temp < *energy)
        {
          register float aux = (*energy) * (*energy - temp) * ((float)cdt1);

          if ((aux>1.0e-12f)||(temp>1.0e-12f))  // !!DeBuG!! Make sure the SQRT argument is never <0, and that we never get 0/0 -> NaN when aux=temp=0 !!
          {
         #ifdef USING_CUDA
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) * rsqrtf(aux + aux + temp * temp) * 1.956951306108245e-6f;
             // 1.956951306108245e-6 = 1.0/510998.918f   //  Version using the reciprocal of sqrt in CUDA: faster and more accurate!!
         #else
           pzomc = cgco_SHARED->fj0[*mat + i__*MAX_MATERIALS] * (aux - temp * 510998.918f) / (sqrtf(aux + aux + temp * temp) * 510998.918f);
         #endif

          }
          else
          {
            pzomc = 0.002f;    // !!DeBuG!! Using a rough approximation to a sample value of pzomc found using pure double precision: NOT RIGUROUS! But this code is expected to be used very seldom, only in extreme cases.
            #ifndef USING_CUDA
            if (warning_flag_1<0)
            {
               warning_flag_1 = +1;  // Disable warning, do not show again
               printf("          [... Small numerical precision error detected computing \"pzomc\" in GCOa (this warning will not be repeated).]\n               i__=%d, aux=%.14f, temp=%.14f, pzomc(forced)=%.14f, uico=%.14f, energy=%.7f, cgco_SHARED->fj0=%.14f, mat=%d, cdt1=%.14lf\n", (int)i__, aux, temp, pzomc, cgco_SHARED->uico[*mat+i__*MAX_MATERIALS], *energy, cgco_SHARED->fj0[*mat+i__*MAX_MATERIALS], (int)*mat, cdt1);   // !!DeBuG!!
            }
            #endif                    
          }
          
          temp = pzomc * 1.4142135623731f;
          if (pzomc > 0.0f)
            temp = 0.5f - (temp + 0.70710678118654502f) * (temp + 0.70710678118654502f);   // Calculate exponential argument
          else
            temp = 0.5f - (0.70710678118654502f - temp) * (0.70710678118654502f - temp);

          temp = 0.5f * expf(temp);      // All threads will calculate the expf together
          
          if (pzomc > 0.0f)
            temp = 1.0f - temp;

          s += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * temp;
          rn[i__] = temp;
        }        
      }
    } while( (ranecu(seed)*s0) > (s*(1.0f+tau*(/*ek1=*/(ek3 - ek2 - 1.0f)+tau*(ek2+tau*ek3)))/(ek3*tau*(tau*tau+1.0f))) );  //  ****  Rejection function

    *costh_Compton = 1.0 - cdt1;
        
/*  ****  Target electron shell. */
    for (;;)
    {
      register float temp = s*ranecu(seed);
      float pac = 0.0f;

      int ishell = my_noscco - 1;     // First shell will have number 0
      for (i__ = 0; i__ < (my_noscco-1); i__++)    // !!DeBuG!! Iterate to (my_noscco-1) only: the last oscillator is excited in case all other fail (no point in double checking) ??
      {
        pac += cgco_SHARED->fco[*mat + i__*MAX_MATERIALS] * rn[i__];   // !!DeBuG!! pac[] is calculated on the fly to save registers!
        if (pac > temp)       //  pac[] is calculated on the fly to save registers!  
        {
            ishell = i__;
            break;
        }
      }

    /*  ****  Projected momentum of the target electron. */
      temp = ranecu(seed) * rn[ishell];

      if (temp < 0.5f)
      {
        pzomc = (0.70710678118654502f - sqrtf(0.5f - logf(temp + temp))) / (cgco_SHARED->fj0[*mat + ishell * MAX_MATERIALS] * 1.4142135623731f);
      }
      else
      {
        pzomc = (sqrtf(0.5f - logf(2.0f - 2.0f*temp)) - 0.70710678118654502f) / (cgco_SHARED->fj0[*mat + ishell * MAX_MATERIALS] * 1.4142135623731f);
      }
      if (pzomc < -1.0f)
      {
        continue;      // re-start the loop
      }

  /*  ****  F(EP) rejection. */
      temp = tau * (tau - (*costh_Compton) * 2.f) + 1.f;       // this variable was originally called "xqc"
      
        // af = sqrt( max_value(temp,1.0e-30f) ) * (tau * (tau - *costh_Compton) / max_value(temp,1.0e-30f) + 1.f);  //!!DeBuG!! Make sure the SQRT argument is never <0, and that I don't divide by zero!!

      if (temp>1.0e-20f)   // !!DeBuG!! Make sure the SQRT argument is never <0, and that I don't divide by zero!!
      {
        af = sqrtf(temp) * (tau * (tau - ((float)(*costh_Compton))) / temp + 1.f);
      }
      else
      {
        // When using single precision, it is possible (but very uncommon) to get costh_Compton==1 and tau==1; then temp is 0 and 'af' can not be calculated (0/0 -> nan). Analysing the results obtained using double precision, we found that 'af' would be almost 0 in this situation, with an "average" about ~0.002 (this is just a rough estimation, but using af=0 the value would never be rejected below).

        af = 0.00200f;    // !!DeBuG!!
                
        #ifndef USING_CUDA
        if (warning_flag_2<0)
        {
            warning_flag_2 = +1;  // Disable warning, do not show again
            printf("          [... Small numerical precision error detected computing \"af\" in GCOa (this warning will not be repeated)].\n               xqc=%.14f, af(forced)=%.14f, tau=%.14f, costh_Compton=%.14lf\n", temp, af, tau, *costh_Compton);    // !!DeBuG!!
        }
        #endif
      }

      if (af > 0.0f)
      {
        temp = af * 0.2f + 1.f;    // this variable was originally called "fpzmax"
      }
      else
      {
        temp = 1.f - af * 0.2f;
      }
      
      if ( ranecu(seed)*temp < /*fpz =*/(af * max_value( min_value(pzomc,0.2f) , -0.2f ) + 1.f) )
      {
        break;
      }

    }

/*  ****  Energy of the scattered photon. */
    {
      register float t, b1, b2, temp;
      t = pzomc * pzomc;
      b1 = 1.f - t * tau * tau;
      b2 = 1.f - t * tau * ((float)(*costh_Compton));

      temp = sqrtf( fabsf(b2 * b2 - b1 * (1.0f - t)) );
      
          
      if (pzomc < 0.0f)
         temp *= -1.0f;

      // !Error! energy may increase (slightly) due to inacurate calculation!  !!DeBuG!!
      t = (tau / b1) * (b2 + temp);
      if (t > 1.0f)
      {
        #ifndef USING_CUDA

        #endif      
        #ifndef USING_CUDA
        if (warning_flag_3<0)
        {
            warning_flag_3 = +1;  // Disable warning, do not show again
            printf("\n          [... a Compton event tried to increase the x ray energy due to precision error. Keeping initial energy. (This warning will not be repeated.)]\n               scaling=%.14f, costh_Compton=%.14lf\n", t, *costh_Compton);   // !!DeBuG!!
        }
        #endif
        
        t = 1.0f; // !!DeBuG!! Avoid increasing energy by hand!!! not nice!!
      }

      (*energy) *= t;
       // (*energy) *= (tau / b1) * (b2 + temp);    //  Original PENELOPE code
    }
    
}  // [End subroutine GCOa]



////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
//!  Tally the depose deposited inside each material.
//!  This function is called whenever a particle suffers a Compton or photoelectric
//!  interaction. The energy released in each interaction is added and later in the 
//!  report function the total deposited energy is divided by the total mass of the 
//!  material in the voxelized object to get the dose. This naturally accounts for
//!  multiple densities for voxels with the same material (not all voxels have same mass).
//!  Electrons are not transported in MC-GPU and therefore we are approximating
//!  that the dose is equal to the KERMA (energy released by the photons alone).
//!  This approximation is acceptable when there is electronic equilibrium and 
//!  when the range of the secondary electrons is shorter than the organ size. 
//!
//!  The function uses atomic functions for a thread-safe access to the GPU memory.
//!  We can check if this tally was disabled in the input file checking if the array
//!  materials_dose was allocated in the GPU (disabled if pointer = NULL).
//!
//!
//!       @param[in] Edep   Energy deposited in the interaction
//!       @param[in] material   Current material id number
//!       @param[out] materials_dose   ulonglong2 array storing the mateials dose [in eV/g] and dose^2 (ie, uncertainty).
////////////////////////////////////////////////////////////////////////////////
#ifdef USING_CUDA
__device__
#endif
inline 
void tally_materials_dose(float* Edep, int* material, ulonglong2* materials_dose)
{
      
// !!DeBuG!! The energy can be tallied directly with atomicAdd in global memory or using shared memory first and then global for whole block if too slow. With the initial testing it looks like using global memory is already very fast!

// !!DeBuG!! WARNING: with many histories and few materials the materials_dose integer variables may overflow!! Using double precision floats would be better. Single precision is not good enough because adding small energies to a large counter would give problems.

#ifdef USING_CUDA
  atomicAdd(&materials_dose[*material].x, __float2ull_rn((*Edep)*SCALE_eV) );  // Energy deposited at the material, scaled by the factor SCALE_eV and rounded.
  atomicAdd(&materials_dose[*material].y, __float2ull_rn((*Edep)*(*Edep)) );   // Square of the dose to estimate standard deviation (not using SCALE_eV for std_dev to prevent overflow)
#else
  materials_dose[*material].x += (unsigned long long int)((*Edep)*SCALE_eV + 0.5f);
  materials_dose[*material].y += (unsigned long long int)((*Edep)*(*Edep) + 0.5f);
#endif     
          
  return;
}
